/* 
* Copyright 2007-2012 Computational Electromagnetic Group (CEM), Dept. ECE, UC San Diego.  All rights reserved.
 * Author: Shaojing Li, March 2012
 */
/*
* field_static_scalar_gpu.cu: class definition of Class FieldStaticScalarGpu
*/
#include "field_static_scalar_gpu.h"
#include "direct.h"
#include "error.h"
#include "gpu.h"
#include "memory.h"
#include "memory_gpu.h"
#include "nbodyfast.h"
#include "nufft.h"

namespace NBODYFAST_NS{
FieldStaticScalarGpu :: FieldStaticScalarGpu(class NBODYFAST *n_ptr) : FieldStaticScalar(n_ptr), FieldGpu(n_ptr), Field(n_ptr)
{
	d_src_amp = NULL;
	d_field_amp = NULL;
}

FieldStaticScalarGpu :: ~FieldStaticScalarGpu()
{
	hipError_t _cuda_error;
	for (int _thread_id = 0; _thread_id < nbodyfast->num_devices; _thread_id++)
	{
		int _cur_dev = nbodyfast->device_name[_thread_id];
		hipError_t _cuda_error;

		nbodyfast->gpu->memory_gpu->free_device<FP_TYPE>(_cuda_error, &d_field_amp[_thread_id], _cur_dev);
		nbodyfast->gpu->memory_gpu->free_device<FP_TYPE>(_cuda_error, &d_src_amp[_thread_id], _cur_dev);
	}

	nbodyfast->memory->free_host<FP_TYPE*>(&d_field_amp);
	nbodyfast->memory->free_host<FP_TYPE*>(&d_src_amp);

}

int FieldStaticScalarGpu :: array_alloc_multi_interface()
{
	FieldStaticScalar::array_alloc_multi_interface(); 
	FieldGpu::array_alloc_multi_interface();

	nbodyfast->error->last_error = amp_field_alloc_multi();

	return 0;
}
int FieldStaticScalarGpu :: amp_field_alloc()
{

	FieldStaticScalar::amp_field_alloc();
	
	nbodyfast->memory->alloc_host<FP_TYPE*>(&d_src_amp, nbodyfast->num_devices, "field->d_src_amp");
	nbodyfast->memory->alloc_host<FP_TYPE*>(&d_field_amp, nbodyfast->num_devices, "field->d_field_amp");

	for (int i = 0; i < nbodyfast->num_devices; i++)
	{
		d_src_amp[i] = NULL;
		d_field_amp[i] = NULL;
	}

	if (nbodyfast->multi_device == false) amp_field_alloc_single();
	// similar to coordinate arrays in the Class FieldGpu, source and amplitude arrays for multi GPU calculation are allocated after preprocessing
	return 0;
}

int FieldStaticScalarGpu :: amp_field_alloc_single()
{	
	hipError_t _cuda_error;

	nbodyfast->gpu->memory_gpu->alloc_device<FP_TYPE>(_cuda_error, &d_src_amp[0], nbodyfast->problem_size, "field->d_src_amp[0]", nbodyfast->gpu->dev_list[0].index);
	nbodyfast->gpu->memory_gpu->alloc_device<FP_TYPE>(_cuda_error, &d_field_amp[0], nbodyfast->problem_size, "field->d_field_amp[0]", nbodyfast->gpu->dev_list[0].index);

	return 0;
}
int FieldStaticScalarGpu :: amp_field_alloc_multi()
{
#pragma omp barrier
#pragma omp parallel
	{
		int _thread_id = omp_get_thread_num();
		int _cur_dev = nbodyfast->device_name[_thread_id];
		hipError_t _cuda_error;
		std::stringstream _array_name;
		_array_name <<"nufft->d_src_amp[" << _thread_id << "]";
		{
			nbodyfast->gpu->memory_gpu->alloc_device<FP_TYPE>(_cuda_error, &d_src_amp[_thread_id], nbodyfast->src_size_dev[_thread_id], _array_name.str(), _cur_dev);
		}
		_array_name.str("");
		_array_name <<"nufft->d_field_amp[" << _thread_id << "]";
		{
			nbodyfast->gpu->memory_gpu->alloc_device<FP_TYPE>(_cuda_error, &d_field_amp[_thread_id], nbodyfast->obs_size_dev[_thread_id], _array_name.str(), _cur_dev);
		}
	}
#pragma omp barrier
//
//	nbodyfast->memory->output_allocated_list();
//	nbodyfast->gpu->memory_gpu->output_allocated_list();

	return 0;
}


// Get source amplitudes from outside, the set_src_amp is usually called at every iteration, at the beginning of execution
int FieldStaticScalarGpu :: set_src_amp(double *_charge)
{
	nbodyfast->error->last_error = nbodyfast->multi_device ? set_src_amp_multi(_charge) : set_src_amp_single(_charge);
	return 0;
}
// single GPU calculation goes here
int FieldStaticScalarGpu :: set_src_amp_single(double *_charge)
{
	hipDeviceSynchronize();		
	if (nbodyfast->algo_name == "nufft")
	{
		int *src_trans_idx = nbodyfast->nufft->get_src_trans_idx();
#pragma omp parallel for
		for (int i = 0; i < nbodyfast->problem_size; i++)
		{
			src_amp[i] = _charge[src_trans_idx[i]]; // to copy data for NUFFT, we need src_trans_idx[] as the reordering table
			field_amp[i] = 0.0f;
		}

		// copy source amplitudes to device and set field amplitudes to 0	
		nbodyfast->gpu->memory_gpu->memcpy_host_to_device<FP_TYPE>(d_src_amp[0], src_amp, nbodyfast->problem_size, nbodyfast->device_name[0]);
		nbodyfast->gpu->memory_gpu->memset_device<FP_TYPE>(d_field_amp[0], 0, nbodyfast->problem_size, nbodyfast->device_name[0]); 
		hipDeviceSynchronize();			

		return 0;
	}

	// default source copying for direct methods
	if (nbodyfast->algo_name == "direct")
	{
		for (int i = 0; i < nbodyfast->problem_size; i++)
		{	
			src_amp[i] = FP_TYPE(_charge[i]);
			field_amp[i] = 0.0f;
		}

		nbodyfast->gpu->memory_gpu->memcpy_host_to_device<FP_TYPE>(d_src_amp[0], src_amp, nbodyfast->problem_size, nbodyfast->gpu->dev_list[0].index);
		nbodyfast->gpu->memory_gpu->memset_device<FP_TYPE>(d_field_amp[0], 0, nbodyfast->problem_size, nbodyfast->gpu->dev_list[0].index);
		hipDeviceSynchronize();			

		return 0;

	}
	
	std::cout << "unknown algorithm..." << std::endl;
	exit(0);
	return -1;

}
// multi GPU calculation goes here
int FieldStaticScalarGpu :: set_src_amp_multi(double *_charge)
{
	hipDeviceSynchronize();		
	if (nbodyfast->algo_name == "nufft")
	{
		int **src_trans_idx_dev = nbodyfast->nufft->get_src_trans_idx_dev();
		int *src_trans_idx = nbodyfast->nufft->get_src_trans_idx();
#pragma omp barrier
#pragma omp parallel
		{
			int _thread_id = omp_get_thread_num();
			int _cur_dev = nbodyfast->device_name[_thread_id];
			hipError_t _cuda_error;

			for (int i = 0; i < nbodyfast->src_size_dev[_thread_id]; i++)
			{
				src_amp_dev[_thread_id][i] = _charge[src_trans_idx[src_trans_idx_dev[_thread_id][i]]];

//				//src_amp_dev[_thread_id][i] = src_amp[src_trans_idx_dev[_thread_id][i]];
//				//src_amp_dev[_thread_id][i] = _charge[src_trans_idx_dev[_thread_id][i]];

				field_amp_dev[_thread_id][i] = 0.0f;
			}
			
			// copy source amplitudes to device and set field amplitudes to 0
			nbodyfast->gpu->memory_gpu->memcpy_host_to_device<FP_TYPE>(d_src_amp[_thread_id], src_amp_dev[_thread_id], nbodyfast->src_size_dev[_thread_id], _cur_dev);
			nbodyfast->gpu->memory_gpu->memset_device<FP_TYPE>(d_field_amp[_thread_id], 0, nbodyfast->obs_size_dev[_thread_id], _cur_dev); 
		}
		hipDeviceSynchronize();		
#pragma omp barrier
		return 0;
	}

	// default source copying for direct methods
	if (nbodyfast->algo_name == "direct")
	{
		std::cout << "direct method on multi GPU is not ready yet" << std::endl;
		std::cout << "stopped at \"int FieldStaticScalarGpu :: set_src_amp_single(double *_charge)\" " << std::endl;
		
		exit(0);

		return 0;

	}
	
	std::cout << "unknown algorithm..." << std::endl;
	exit(0);
	return -1;

}

// Transfer field amplitudes to outside, the set_fld_amp is usually called at every iteration, at the end of execution. They are symetrical to set_src series of subroutines
int FieldStaticScalarGpu :: set_fld_amp(double *_field)
{
	nbodyfast->error->last_error = nbodyfast->multi_device ? set_fld_amp_multi(_field) : set_fld_amp_single(_field);
	return 0;
}
// single GPU calculation goes here
int FieldStaticScalarGpu :: set_fld_amp_single(double *_field)
{
	hipDeviceSynchronize();		
	if (nbodyfast->algo_name == "nufft")
	{
		nbodyfast->gpu->memory_gpu->memcpy_device_to_host<FP_TYPE>(field_amp, d_field_amp[0], nbodyfast->problem_size, nbodyfast->gpu->dev_list[0].index);
		int *obs_trans_idx = nbodyfast->nufft->get_obs_trans_idx();
#pragma omp parallel for
		for (int i = 0; i < nbodyfast->problem_size; i++)
		{
			_field[obs_trans_idx[i]] = field_amp[i];
		}	
		hipDeviceSynchronize();		
		return 0;
	}

	// default source copying for direct methods
	if (nbodyfast->algo_name == "direct")
	{
		nbodyfast->gpu->memory_gpu->memcpy_device_to_host<FP_TYPE>(field_amp, d_field_amp[0], nbodyfast->problem_size, nbodyfast->gpu->dev_list[0].index);

		for (int i = 0; i < nbodyfast->problem_size; i++)
		{	
			_field[i] = double(field_amp[i]);
			src_amp[i] = 0.0f;
			field_amp[i] = 0.0f;
		}

		//hipDeviceSynchronize();			

		return 0;
	}
	
	std::cout << "unknown algorithm..." << std::endl;
	exit(0);
	return -1;
}
// multi GPU calculation goes here
int FieldStaticScalarGpu :: set_fld_amp_multi(double *_field)
{
	hipDeviceSynchronize();		
	if (nbodyfast->algo_name == "nufft")
	{
		int **obs_trans_idx_dev = nbodyfast->nufft->get_obs_trans_idx_dev();
		int *obs_trans_idx = nbodyfast->nufft->get_obs_trans_idx();
#pragma omp barrier
#pragma omp parallel
		{

			int _thread_id = omp_get_thread_num();
			int _cur_dev = nbodyfast->device_name[_thread_id];
			hipError_t _cuda_error;
			nbodyfast->gpu->memory_gpu->memcpy_device_to_host<FP_TYPE>(field_amp_dev[_thread_id], d_field_amp[_thread_id], nbodyfast->obs_size_dev[_thread_id], _cur_dev);

			for (int i = 0; i < nbodyfast->obs_size_act_dev[_thread_id]; i++)
			{
				//field_amp[obs_trans_idx_dev[_thread_id][i]] = field_amp_dev[_thread_id][i];
				_field[obs_trans_idx[obs_trans_idx_dev[_thread_id][i]]] = field_amp_dev[_thread_id][i];
				//_field[obs_trans_idx_dev[_thread_id][i]] = field_amp_dev[_thread_id][i];

			}
			//hipDeviceSynchronize();			
		}
//#pragma omp barrier
//#pragma omp parallel for
//		for (int i = 0; i < problem_size; i++)
//		{
//			_field[obs_trans_idx[i]] = field_amp[i];
//		}
#pragma omp barrier
		hipDeviceSynchronize();		
		return 0;
	}

	// default source copying for direct methods
	if (nbodyfast->algo_name == "direct")
	{
		std::cout << "direct method on multi GPU is not ready yet" << std::endl;
		std::cout << "stopped at \"int FieldStaticScalarGpu :: set_src_amp_single(double *_charge)\" " << std::endl;
		
		exit(0);

		return 0;

	}
	
	std::cout << "unknown algorithm..." << std::endl;
	exit(0);
	return -1;
}


}
